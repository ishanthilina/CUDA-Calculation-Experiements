// clear;rm a.out; nvcc -O3 q3.cu ;./a.out -c
// 
// clear;rm a.out; nvcc -O3 -D DP -L /usr/local/cuda/lib -lcuda q3.cu ;./a.out -c

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <pthread.h>

#define MATRIX_DIM 1800

#define MIN_ERROR 0.1

// CUDA related
#define BLOCK_SIZE 32


//Code to check for GPU errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code),\
		 file, line);
		if (abort) exit(code);
	}
}

//Help code for switching between Single Precision and Double Precision
#ifdef DP
typedef double Real;
#else
typedef float Real;
#endif


static unsigned long inKB(unsigned long bytes)

{ return bytes/1024; }



static unsigned long inMB(unsigned long bytes)

{ return bytes/(1024*1024); }


/**
 * Used to print memory states in the GPU
 */
 static void printStats()

 {

 	size_t free, total;

 	hipError_t res = hipMemGetInfo(&free, &total);

 	if(res != hipSuccess){
 		printf("!!!! hipMemGetInfo failed! (status = %x)", res);
 		return;

 	}

 	printf("---------------------------------------------------------------\n");

 	printf("^^^^ Free : %lu bytes (%lu KB) (%lu MB)\n", free, inKB(free), \
 		inMB(free));

 	printf("^^^^ Total: %lu bytes (%lu KB) (%lu MB)\n", total, inKB(total), \
 		inMB(total));

 	printf("^^^^ %f%% free, %f%% used\n", 100.0*free/(double)total, \
 		100.0*(total - free)/(double)total);
 	printf("---------------------------------------------------------------\n");

 }

/**
 * Carries out a simple square matrix multiplication where each thread
 * calculates a single item in the resulting matrix.
 * @param A First matrix
 * @param B Second matrix
 * @param C Results matrix
 */
 __global__ void cuda_simple_mat_mul(Real* A, Real* B, Real* C) {

 	int col = threadIdx.x + blockIdx.x * blockDim.x;
 	int row = threadIdx.y + blockIdx.y * blockDim.y;

	//check for bounds
 	if(row < MATRIX_DIM && col < MATRIX_DIM)
 	{
 		Real sum = 0.f;

 		for (int i = 0; i < MATRIX_DIM; i++)
 			sum += A[row * MATRIX_DIM + i] * B[i * MATRIX_DIM + col];

 		C[row * MATRIX_DIM + col] = sum;
 	}
 }

/**
 * Initializes the given matrix to a set of float/Double values between 1-2
 */
 void init_matrix(Real matrix[MATRIX_DIM][MATRIX_DIM])
 {
 	for(int i=0; i < MATRIX_DIM; i++)
 	{
 		for(int j=0; j < MATRIX_DIM; j++)
 		{
 			matrix[i][j] = 1 + (Real)rand()/(Real)RAND_MAX;
 		}
 	}
 }

/**
 * Prints the given matrix to the stdout
 */
 void print_matrix(Real matrix[MATRIX_DIM][MATRIX_DIM])
 {

 	for(int i = 0; i < MATRIX_DIM; i++)
 	{
 		printf("[");
 			for(int j  = 0; j < MATRIX_DIM; j++)
 			{
		#ifdef DP
 				printf("%20.18f ", matrix[i][j]);
    	#else
 				printf("%f ", matrix[i][j]);
    	#endif

 				
 			}
 			printf("] \n");
 		}
 		printf("\n");
 	}

/**
 * Compares the given two matrices.
 */
 void compare_matrices(Real matrix1[MATRIX_DIM][MATRIX_DIM],\
 	Real matrix2[MATRIX_DIM][MATRIX_DIM])
 {
 	for(int i = 0; i < MATRIX_DIM; i++)
 	{
 		for(int j = 0; j < MATRIX_DIM; j++)
 		{
 			if((matrix1[i][j] - matrix2[i][j] > MIN_ERROR) &&
 				(matrix1[i][j] - matrix2[i][j] > 0))
 			{
 				printf("Error i=%d : j=%d mat1=%f mat2=%f\n",i,j,\
 					matrix1[i][j], matrix2[i][j]);
 				return;
 			}
 		}
 	}

 	printf("Matrices Match! \n");
 } 
/**
 * carries out a serial matrix multiplication
 */
 void serial_mat_mul(Real A[MATRIX_DIM][MATRIX_DIM], \
 	Real B[MATRIX_DIM][MATRIX_DIM], Real C[MATRIX_DIM][MATRIX_DIM])	{
 	float sum;
 	for (int row=0; row<MATRIX_DIM; row++){
 		for (int col=0; col<MATRIX_DIM; col++){
 			sum = 0.f;
 			for (int n=0; n<MATRIX_DIM; n++){
 				sum += A[row][n]*B[n][col];
 			}
 			C[row][col] = sum;
 		}
 	}
 }

/**
 * Shows the usage of the program.
 */
 void print_usage(){
 	printf("Wrong usage!\n");
 }

/**
 * Does a matrix multiplication using the "tiled" approach in the GPU
 * @param A First matrix
 * @param B Second matrix
 * @param C Results matrix
 */
 __global__ void cuda_tiled_mat_mul(Real * A, Real * B, Real * C) {
 	
 	float CValue = 0;

 	int Row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
 	int Col = blockIdx.x*BLOCK_SIZE + threadIdx.x;

 	__shared__ Real As[BLOCK_SIZE][BLOCK_SIZE];
 	__shared__ Real Bs[BLOCK_SIZE][BLOCK_SIZE];

 	for (int k = 0; k < (BLOCK_SIZE + MATRIX_DIM - 1)/BLOCK_SIZE; k++) {
 		// check ranges for the matrices and check for left out parts where
 		//  MATRIX_DIM is not an exact multiplication of tile size(BLOCK_SIZE)
 		if (k*BLOCK_SIZE + threadIdx.x < MATRIX_DIM && Row < MATRIX_DIM){

 			As[threadIdx.y][threadIdx.x] = A[Row*MATRIX_DIM + \
 				k*BLOCK_SIZE + threadIdx.x];
 		}  
 		else{

 			As[threadIdx.y][threadIdx.x] = 0.0;
 		}                                                   

 		if (k*BLOCK_SIZE + threadIdx.y < MATRIX_DIM && Col < MATRIX_DIM){
 			
 			Bs[threadIdx.y][threadIdx.x] = B[(k*BLOCK_SIZE + \
 				threadIdx.y)*MATRIX_DIM + Col];
 		}
 		else{

 			Bs[threadIdx.y][threadIdx.x] = 0.0;
 		}                                                   

 		// Wait till all the threads finish before calculating the results
 		__syncthreads();

 		for (int n = 0; n < BLOCK_SIZE; ++n) 
 			CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

 		__syncthreads();
 	}

 	// Calculate the result
 	if (Row < MATRIX_DIM && Col < MATRIX_DIM) 
 		C[((blockIdx.y * blockDim.y + threadIdx.y)*MATRIX_DIM)+\
 			(blockIdx.x*blockDim.x)+threadIdx.x]=CValue;

 }


 int main(int argc, char const *argv[])
 {

 	if(argc<2){
 		print_usage();
 	}

 	// Initialize the random seed
 	srand(time(NULL));

 	// Create the matrices
 	static Real A[MATRIX_DIM][MATRIX_DIM]; 
 	static Real B[MATRIX_DIM][MATRIX_DIM]; 
 	static Real C[MATRIX_DIM][MATRIX_DIM]; 
 	static Real serial_C[MATRIX_DIM][MATRIX_DIM]; 
 	// Initialize the matrices
 	init_matrix(A);
 	init_matrix(B);
 	// print_matrix(A);
 	// print_matrix(B);


 	if (0 == strcmp(argv[1], "-s"))
 	{
 		printf("serial mode\n");
 	}
 	else if (0 == strcmp(argv[1], "-p"))
 	{
 		printf("pthread mode\n");
 	}
 	else if (0 == strcmp(argv[1], "-c"))
 	{

 		long matrix_size=MATRIX_DIM*MATRIX_DIM*sizeof(Real);
 			// printf("%ld\n",matrix_size );

 		Real* _A;
 		gpuErrchk(hipMalloc((void**) &_A, matrix_size));
 		// printStats();

 		Real* _B;
 		gpuErrchk(hipMalloc((void**) &_B, matrix_size));
 		// printStats();

 		Real* _C;
 		gpuErrchk(hipMalloc((void**) &_C, matrix_size));
 		// printStats();

 		// copy the matrices to device
 		hipMemcpy(_A, A, matrix_size, hipMemcpyHostToDevice);
 		hipMemcpy(_B, B, matrix_size, hipMemcpyHostToDevice);

 		// If the tiled mode needs to be enabled
 		if (argc>2 && 0 == strcmp(argv[2], "-t")){
 			printf("cuda tiled mode\n");

 			// set the grid and block sizes
 			dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
 			dim3 dimGrid;
 			dimGrid.x = (MATRIX_DIM + dimBlock.x - 1)/dimBlock.x;
 			dimGrid.y = (MATRIX_DIM + dimBlock.y - 1)/dimBlock.y;

 			// execute the workload in the GPU
 			cuda_tiled_mat_mul<<<dimGrid , dimBlock>>>(_A,_B,_C);

 			// Copy back the result
 			hipMemcpy(C,_C,matrix_size,hipMemcpyDeviceToHost);

 			// get the serial output
 			serial_mat_mul(A,B,serial_C);

 			// print_matrix(serial_C);
 			// print_matrix(C);

 			// Compare the reuslts
 			compare_matrices(serial_C,C);

 			// free device memory
 			hipFree(_A);
 			hipFree(_B);
 			hipFree(_C);

 		}
 		else{
 			printf("cuda mode\n");

 			int K=100;			
 			
 			dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
 			dim3 grid(K,K);

 			// call the GPU
 			cuda_simple_mat_mul<<<grid,threadBlock>>>(_A,_B,_C);

 			// Copy back the result
 			hipMemcpy(C,_C,matrix_size,hipMemcpyDeviceToHost);

 			// get the serial output
 			serial_mat_mul(A,B,serial_C);

 			// print_matrix(serial_C);
 			// print_matrix(C);

 			compare_matrices(serial_C,C);

 			// free device memory
 			hipFree(_A);
 			hipFree(_B);
 			hipFree(_C);


 		}
 		
 	}
 	else{
 		print_usage();
 	}
 	return 0;
 }
