#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <pthread.h>

//Help code for switching between Single Precision and Double Precision
#ifdef DP
typedef double Real;
#else
typedef float Real;
#endif