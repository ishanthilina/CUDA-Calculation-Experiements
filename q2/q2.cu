// clear;rm a.out; nvcc -O3 -D DP -L /usr/local/cuda/lib -lcuda -arch sm_30 q2.cu ;./a.out -c
// 
// clear;rm a.out; nvcc -O3 -L /usr/local/cuda/lib -lcuda q2.cu ;./a.out -c
// 
// 
// http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <pthread.h>
#include <hip/hip_runtime_api.h>


// CUDA related
#define THREADS_PER_BLOCK 256
#define CALCS_PER_THREAD 50

// PThread related
#define MAX_PTHREADS 8

#define VECTOR_SIZE 10000000  //1e8
// #define VECTOR_SIZE 5  //1e8

//Code to check for GPU errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
 if (code != hipSuccess) 
 {
  fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
  if (abort) exit(code);
}
}
//Help code for switching between Single Precision and Double Precision
#ifdef DP
typedef double Real;
#else
typedef float Real;
#endif

/**
 * Shows the usage of the program.
 */
 void print_usage(){
 	printf("Wrong usage!\n");
 }

 void print_vector(Real vector[]){
 	printf("---------------------------------------------------------------\n");
 	for(long i=0;i<VECTOR_SIZE;i++){
		#ifdef DP
 		printf("%ld --> %20.18f\n",i,vector[i]);

        #else
 		printf("%ld --> %f\n",i,vector[i]);


        #endif
 	}
 	printf("---------------------------------------------------------------\n");
 }

 static unsigned long inKB(unsigned long bytes)

 { return bytes/1024; }



 static unsigned long inMB(unsigned long bytes)

 { return bytes/(1024*1024); }


/**
 * Used to print memory states in the GPU
 */
 static void printStats()

 {

  size_t free, total;

  hipError_t res = hipMemGetInfo(&free, &total);

  if(res != hipSuccess){
   printf("!!!! hipMemGetInfo failed! (status = %x)", res);
   return;

 }

 printf("---------------------------------------------------------------\n");

 printf("^^^^ Free : %lu bytes (%lu KB) (%lu MB)\n", free, inKB(free), inMB(free));

 printf("^^^^ Total: %lu bytes (%lu KB) (%lu MB)\n", total, inKB(total), inMB(total));

 printf("^^^^ %f%% free, %f%% used\n", 100.0*free/(double)total, 100.0*(total - free)/(double)total);
 printf("---------------------------------------------------------------\n");

}


/**
 * Initializes a given vector with values between 1 and 2
 * @param vector The vector that needs to be initialized
 */
 void initialize_vector(Real vector[]){
 	for(long i=0;i<VECTOR_SIZE;i++){
 		vector[i]=(rand() / (float) RAND_MAX)+1;
 	}
 }

/**
 * Does a serial calculation of the dot product of the two given vectors
 * @param  vector1 
 * @param  vector2 
 * @return Dot product value of the vectors
 */
 Real serial_calculation(Real vector1[], Real vector2[]){

 	Real result;

 	for(long i=0;i<VECTOR_SIZE;i++){
 		result += vector1[i] * vector2[i];
 	}

 	return result;
 }

/**
 * prints the value of pi
 * @param pi   calculated value for pi
 * @param from the name of the method that the pi value was calculated. ex- CUDA
 */
 void print_product(Real pi, char *from){
	#ifdef DP
 	char *latter=" result is %20.18f\n";
    #else
 	char *latter=" result is %f\n";
    #endif

 	char *to_print = (char *)malloc(\
			strlen(from)+strlen(latter)+1);//+1 for 0-terminator
 	strcpy(to_print, from);
 	strcat(to_print, latter);
 	printf(to_print,pi);

    // free memory
 	free(to_print);


 }

//struct for parameter passing between pthread calls
 struct pthread_arg_struct {
 	int tid;
 	int total_threads;
 	Real *vector1;
 	Real *vector2;
 };


 void *pthread_calculation(void *arguments){

 	struct pthread_arg_struct *args = (struct pthread_arg_struct *)arguments;
 	int total_threads = args -> total_threads;
	int tid = args -> tid;       //obtain the value of thread id
	Real *vector1=args -> vector1;
	Real *vector2=args -> vector2;

	Real *result = (Real *)malloc(sizeof(Real));
	*result=0;

	// printf("%d\n",tid );
	// print_vector(vector1);
	// print_vector(vector2);

	// calculate the range to be multiplied
	int chunk_size = VECTOR_SIZE/total_threads;
	int lowerbound=chunk_size*tid;			// lowest index to be calculated
	int upperbound=lowerbound+chunk_size-1;	// highest index to be calculated
	// printf("%d calculates from %d to %d\n",tid, lowerbound,upperbound );

	for(int i=lowerbound;i<=upperbound;i++){
		*result+=vector1[i]*vector2[i];
	}

	// allocate the leftover vector elements to master
	if(0==tid && (0!=VECTOR_SIZE%total_threads)){
		for(int i=chunk_size*total_threads;i<=VECTOR_SIZE;i++){
			*result+=vector1[i]*vector2[i];
		}

	}

	// printf("In the end thread %d total is %f\n",tid,*result );
   	pthread_exit((void *)result);     //return the in count

   }

/**
 * Vector dot product code for a single CUDA thread. Function assumes that 
 * the VECTOR_SIZE is completely divisible by CALCS_PER_THREAD
 * 
 * @param vector1 First vector - An array of Real
 * @param vector2 Second vector - An array of Real
 * @param result  Used to return the result - An array of Real
 */
 __global__ void cuda_thread_task(Real *vector1, Real *vector2, Real *result) {


 	unsigned long start_point = threadIdx.x + blockDim.x * blockIdx.x;


	// calculate the range to be multiplied
 	long lowerbound=start_point*CALCS_PER_THREAD;
 	
 	long upperbound=lowerbound+CALCS_PER_THREAD-1;
  // long i=0;
	// printf("%ld - %ld - %ld \n", start_point,lowerbound,upperbound);
  for(long index=lowerbound;index<=upperbound;index++){
 		// printf("%ld - %ld - %ld - %ld\n", start_point,lowerbound,upperbound,index);
    result[index] = vector1[index]*vector2[index];
 		// result[1] = 13;
      // printf("%d - %f\n", threadIdx.x,result[index]);
    // i=index;
 		// vector1[index]=12;

  }


  // printf("%ld -- %ld ::  %f\n", start_point,i,result[i]);

 		// printf("2-Hello thread %d\n", threadIdx.x);
 	// result[start_point]=23;




}

int main(int argc, char const *argv[])
{
	// check the inputs and set the mode
	// int execution_mode=-1;
  if(argc<2){
   print_usage();
 }
	// initialize the vectors
	// printf("%d\n",VECTOR_SIZE);
 static Real vector1[VECTOR_SIZE];
 static Real vector2[VECTOR_SIZE];
 initialize_vector(vector1);
 initialize_vector(vector2);

	// print_vector(vector1);
	// print_vector(vector2);

	// if a serial execution is needed
 if(0==strcmp(argv[1],"-s")){
   printf("serial mode\n");
		// printf("%f\n",serial_calculation(vector1,vector2) );
   print_product(serial_calculation(vector1,vector2),"SERIAL");
 }
	// if a parallel execution is needed
 else if(0==strcmp(argv[1],"-p")){
   print_product(serial_calculation(vector1,vector2),"SERIAL");

   printf("parallel mode\n");

   int num_of_threads;
		// check whether the given # of threads is valid
   if(argc !=3){
    print_usage();
    return -1;
  }
  num_of_threads=atoi(argv[2]);
  if(num_of_threads>MAX_PTHREADS){
    printf("[ERROR-PTHREADS] - Only up to 8 threads can be created\n");
    return -1;
  }

		// printf("Creating %d threads\n", num_of_threads);
  pthread_t threads[num_of_threads];
  int rc;
  long t;
  void *status;
  Real result=0;

   		//initialize the threads
  for(t=0;t<num_of_threads;t++){
    struct pthread_arg_struct* args=(\
     struct pthread_arg_struct*)malloc(sizeof *args);

    args->total_threads=num_of_threads;
    args->tid=t;
    args-> vector1=vector1;
    args-> vector2=vector2;

    rc = pthread_create(&threads[t], NULL, pthread_calculation,(void *)args);
    if (rc){
     printf("ERROR; return code from pthread_create() is %d\n", rc);
     exit(-1);
   }
 }

   		//join the threads
 for(t=0;t<num_of_threads;t++){
  pthread_join(threads[t], &status);
            result+=*(Real*)status;            //keep track of the total in count
            // printf("Thread: %ld %f\n",t,result );

          }

          print_product(result,"PTHREADS");

        }
	// if CUDA execution is needed
        else if(0==strcmp(argv[1],"-c")){
         print_product(serial_calculation(vector1,vector2),"SERIAL");

         printf("cuda mode\n");

         

		//Allocate vectors in device memory
         // printStats();
         size_t size = VECTOR_SIZE * sizeof(Real);
         Real* _vector1;
         gpuErrchk(hipMalloc((void**) &_vector1, size));
         printStats();

         Real* _vector2;
         gpuErrchk(hipMalloc((void**) &_vector2, size));

         printStats();

         Real* _results;
         gpuErrchk(hipMalloc((void**) &_results, size));
         printStats();
         

		// Allocate memory for results in the host memory
    	// Real* results = (Real*)malloc(size);
         static Real results[VECTOR_SIZE]; 

		//copy vectors from host memory to device memory
         hipMemcpy(_vector1, vector1,size,hipMemcpyHostToDevice);
         hipMemcpy(_vector2, vector2,size,hipMemcpyHostToDevice);
    	// cudaMemcpy(_results, results,size,cudaMemcpyHostToDevice);


         long num_of_grids=(VECTOR_SIZE/(THREADS_PER_BLOCK*CALCS_PER_THREAD))+1;
         printf("#of Grids = %ld\n",num_of_grids );
		// carry out the calculations
         cuda_thread_task\
         <<<num_of_grids,THREADS_PER_BLOCK>>>(_vector1,_vector2,_results);

         // gpuErrchk( cudaPeekAtLastError() );
         // gpuErrchk( cudaDeviceSynchronize() );

		// copy the results back from the device memory to host memory
         hipMemcpy(results,_results,size,hipMemcpyDeviceToHost);

		// free device memory
         hipFree(_vector1);
         hipFree(_vector2);
         hipFree(_results);


		// calculate the final result
         Real result=0;
         for(long i=0;i<VECTOR_SIZE;i++){
          result+=results[i];
    		// if(results[i]!=0.0){
    		// 	printf("%f\n",results[i] );

    		// }
        }

        print_product(result,"CUDA");


      }
      else{
       print_usage();
     }
     return 0;
   }